#include "hip/hip_runtime.h"
//Optimized using shared memory and on chip memory 																																			
// nvcc microPlastics.cu -o microPlastics -lglut -lm -lGLU -lGL
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

FILE* ffmpeg;

#define BOLD_ON  "\e[1m"
#define BOLD_OFF   "\e[m"

#define PI 3.141592654
#define BLOCK 256

FILE* MovieFile;
int* Buffer;
int MovieFlag; // 0 movie off, 1 movie on

// Globals to be read in from parameter file.
int NumberOfMicroPlastics;
double DensityOfMicroPlasticMin;
double DensityOfMicroPlasticMax;
float DiameterOfMicroPlasticMin;
float DiameterOfMicroPlasticMax;

int NumberOfPolymerChains;
int PolymersChainLengthMin;
int PolymersChainLengthMax;

float PolymersConnectionLength;
double DensityOfPolymer;
float DiameterOfPolymer;

float BeakerRadius; //4900.0;
float FluidHeight; //118000.0;

float FluidDensity;
float Drag;

float TotalRunTime;
float Dt;
int DrawRate;
int PrintRate;

float PolymerRed;
float PolymerGreen;
float PolymerBlue;

float MicroPlasticRed;
float MicroPlasticGreen;
float MicroPlasticBlue;

// Other Globals
int Pause;
int ViewFlag; // 0 orthoganal, 1 fulstum
int NumberOfBodies;
int NumberOfPolymers;
float4 *BodyPosition, *BodyVelocity, *BodyForce;
float4 *BodyPositionGPU, *BodyVelocityGPU, *BodyForceGPU;
int *PolymerChainLength;
int *PolymerConnectionA, *PolymerConnectionB;
int *PolymerConnectionAGPU, *PolymerConnectionBGPU;
hiprandState_t* DevStates;
dim3 Blocks, Grids;
int DrawTimer, PrintTimer;
float RunTime;
float4 CenterOfSimulation;
float4 AngleOfSimulation;

int DebugFlag;
int RadialConfinementViewingAids;
int StirFlag;
float StirAngularVelosity;
float Theta;
int ShakeItUpFlag;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize;
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

// Prototyping functions
void readSimulationParameters();
void setNumberOfBodies();
void allocateMemory();
void setInitailConditions();
void drawPicture();
void nBody();
void errorCheck(const char*);
void terminalPrint();
void setup();

__global__ void init_curand(unsigned int, hiprandState_t*);
__device__ float4 brownian_motion(hiprandState_t*, int);
__device__ float4 shakeItUp(hiprandState_t*, int);
__device__ float4 getPolymerPolymerForce(float4 , float4 , int , int , int , float, int );
__device__ float4 getPolymerMicroPlasticForce(float4 , float4 );
__device__ float4 getMicroPlasticMicroPlasticForce(float4 , float4 );
__device__ float4 getGravityForces(float , float , float );
__device__ float4 getContainerForces(float4 , float , float );
__device__ float4 getStirringForces(hiprandState_t* , int , float4 , float4 , float , float , float );
__global__ void getForces(hiprandState_t* , float4 *, float4 *, float4 *, int *, int *, float , int , int , float , float , float , int , float , int );
__global__ void getForcesSetup(hiprandState_t* , float4 *, float4 *, float4 *, int *, int *, float , int , int , float , float , float , int , float , int );
__global__ void moveBodies(float4 *pos, float4 *, float4 *, float , float , int);

#include "./callBackFunctions.h"

void readSimulationParameters()
{
	ifstream data;
	string name;
	
	data.open("./simulationSetup");
	
	if(data.is_open() == 1)
	{
		getline(data,name,'=');
		data >> NumberOfMicroPlastics;
		
		getline(data,name,'=');
		data >> DensityOfMicroPlasticMin;

		getline(data,name,'=');
		data >> DensityOfMicroPlasticMax;
		
		getline(data,name,'=');
		data >> DiameterOfMicroPlasticMin;

		getline(data,name,'=');
		data >> DiameterOfMicroPlasticMax;
		
		getline(data,name,'=');
		data >> NumberOfPolymerChains;
		
		getline(data,name,'=');
		data >> PolymersChainLengthMin;

		getline(data,name,'=');
		data >> PolymersChainLengthMax;
		
		getline(data,name,'=');
		data >> PolymersConnectionLength;
		
		getline(data,name,'=');
		data >> DensityOfPolymer;
		
		getline(data,name,'=');
		data >> DiameterOfPolymer;
		
		getline(data,name,'=');
		data >> BeakerRadius;

		getline(data,name,'=');
		data >> FluidHeight;

		getline(data,name,'=');
		data >> FluidDensity;

		getline(data,name,'=');
		data >> Drag;
		
		getline(data,name,'=');
		data >> TotalRunTime;
		
		getline(data,name,'=');
		data >> Dt;
		
		getline(data,name,'=');
		data >> DrawRate;
		
		getline(data,name,'=');
		data >> PrintRate;

		getline(data,name,'=');
		data >> PolymerRed;

		getline(data,name,'=');
		data >> PolymerGreen;

		getline(data,name,'=');
		data >> PolymerBlue;

		getline(data,name,'=');
		data >> MicroPlasticRed;

		getline(data,name,'=');
		data >> MicroPlasticGreen;

		getline(data,name,'=');
		data >> MicroPlasticBlue;
		
	}
	else
	{
		printf("\nTSU Error could not open simulationSetup file\n");
		exit(0);
	}
	data.close();
	
	if(DebugFlag == 1)
	{
	//prinf all the parameters
		printf("\n\n Number of MicroPlastics = %d", NumberOfMicroPlastics);
		printf("\n DensityOfMicroPlasticMin = %f", DensityOfMicroPlasticMin);
		printf("\n DensityOfMicroPlasticMax = %f", DensityOfMicroPlasticMax);
		printf("\n DiameterOfMicroPlasticMin = %f", DiameterOfMicroPlasticMin);
		printf("\n DiameterOfMicroPlasticMax = %f", DiameterOfMicroPlasticMax);
		printf("\n NumberOfPolymerChains = %d", NumberOfPolymerChains);
		printf("\n PolymersChainLengthMin = %d", PolymersChainLengthMin);
		printf("\n PolymersChainLengthMax = %d", PolymersChainLengthMax);
		printf("\n PolymersConnectionLength = %f", PolymersConnectionLength);
		printf("\n DensityOfPolymer = %f", DensityOfPolymer);
		printf("\n DiameterOfPolymer = %f", DiameterOfPolymer);
		printf("\n BeakerRadius = %f", BeakerRadius);
		printf("\n FluidHeight = %f", FluidHeight);
		printf("\n FluidDensity = %f", FluidDensity);
		printf("\n Drag = %f", Drag);
		printf("\n TotalRunTime = %f", TotalRunTime);
		printf("\n Dt = %f", Dt);
		printf("\n DrawRate = %d", DrawRate);
		printf("\n PrintRate = %d", PrintRate);
		printf("\n PolymerRed = %f", PolymerRed);
		printf("\n PolymerGreen = %f", PolymerGreen);
		printf("\n PolymerBlue = %f", PolymerBlue);
		printf("\n MicroPlasticRed = %f", MicroPlasticRed);
		printf("\n MicroPlasticGreen = %f", MicroPlasticGreen);
		printf("\n MicroPlasticBlue = %f", MicroPlasticBlue);
	}
	printf("\n\n Parameter file has been read");
	printf("\n");
}

void setNumberOfBodies()
{
	time_t t;
	
	PolymerChainLength = (int*)malloc(NumberOfPolymerChains*sizeof(int));
	
	srand((unsigned) time(&t));
	for(int i = 0; i < NumberOfPolymerChains; i++)
	{
		PolymerChainLength[i] = ((float)rand()/(float)RAND_MAX)*(PolymersChainLengthMax - PolymersChainLengthMin) + PolymersChainLengthMin;
		//printf("\n PolymerChainLength[%d] = %d", i, PolymerChainLength[i]);	
	}
	
	NumberOfPolymers = 0;
	for(int i = 0; i < NumberOfPolymerChains; i++)
	{
		NumberOfPolymers += PolymerChainLength[i];	
	}
	
	NumberOfBodies = NumberOfMicroPlastics + NumberOfPolymers;
	
	if(DebugFlag == 1)
	{
		printf("\n\n Number of Polymers = %d", NumberOfPolymers);
		printf("\n Number of MicroPlastics = %d", NumberOfMicroPlastics);
		printf("\n Total number of bodies = %d", NumberOfBodies);
	}
	
	printf("\n\n Number of bodies has been set");
}

void allocateMemory()
{
	Blocks.x = BLOCK;
	Blocks.y = 1;
	Blocks.z = 1;
	
	Grids.x = (NumberOfBodies - 1)/Blocks.x + 1;
	Grids.y = 1;
	Grids.z = 1;
	
	BodyPosition = (float4*)malloc(NumberOfBodies*sizeof(float4));
	BodyVelocity = (float4*)malloc(NumberOfBodies*sizeof(float4));
	BodyForce    = (float4*)malloc(NumberOfBodies*sizeof(float4));
	
	PolymerConnectionA    = (int*)malloc(NumberOfPolymers*sizeof(int));
	PolymerConnectionB    = (int*)malloc(NumberOfPolymers*sizeof(int));
	
	hipMalloc( (void**)&BodyPositionGPU, NumberOfBodies *sizeof(float4));
	errorCheck("hipMalloc BodyPositionGPU");
	hipMalloc( (void**)&BodyVelocityGPU, NumberOfBodies *sizeof(float4));
	errorCheck("hipMalloc BodyDiameterOfBodyVelocityGPU");
	hipMalloc( (void**)&BodyForceGPU, NumberOfBodies *sizeof(float4));
	errorCheck("hipMalloc BodyForceGPU");
	
	hipMalloc( (void**)&PolymerConnectionAGPU, NumberOfPolymers *sizeof(int));
	errorCheck("hipMalloc BodyForceGPU");
	hipMalloc( (void**)&PolymerConnectionBGPU, NumberOfPolymers *sizeof(int));
	errorCheck("hipMalloc BodyForceGPU");
	
	hipMalloc((void**)&DevStates, NumberOfBodies * sizeof(hiprandState_t));
	
	printf("\n\n Memory has been allocated");
	printf("\n");
}

void polymerShakeUp(float4 *pos, float4 *vel, float4 *force, int *linkA, int *linkB, float length, int n, float drag, float dt, float beakerRadius, float fluidHeight)
{
	float dx, dy, dz, r2, r;
	float penitration;
	float k1 = 100.0;
	float k2 = 100.0;
	float forceMag;
	float magx, magy, magz, mag;
	float dragTemp;
	float k = 100.0;
	
	mag = 10.0;
	float stopTime = 2.0;
	float time = 0;
	DrawTimer = 0;
	
	hipMemcpy( PolymerConnectionAGPU, PolymerConnectionA, NumberOfPolymers*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( PolymerConnectionBGPU, PolymerConnectionB, NumberOfPolymers*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( BodyPositionGPU, BodyPosition, NumberOfBodies*sizeof(float4), hipMemcpyHostToDevice );
	hipMemcpy( BodyVelocityGPU, BodyVelocity, NumberOfBodies*sizeof(float4), hipMemcpyHostToDevice );
	hipMemcpy( BodyForceGPU, BodyForce, NumberOfBodies*sizeof(float4), hipMemcpyHostToDevice );
		
	while(time < stopTime)
	{
		for(int i = 0; i < n; i++)
		{
			force[i].x = 0.0;
			force[i].y = 0.0;
			force[i].z = 0.0;
		}
		
		for(int i = 0; i < n; i++)
		{
			mag = 10.0;
			if(time < 1.0)
			{
				dragTemp = 0.001;
				if(linkA[i] == -1) 
				{
					force[i].y -= 500.0;
					force[i].x -= 10.0;
				}
				if(linkB[i] == -1) 
				{
					force[i].y += 500.0;
					force[i].x += 10.0;
				}
				
				magx = mag*((float)rand()/RAND_MAX*2.0 - 1.0);
				magy = mag*((float)rand()/RAND_MAX*2.0 - 1.0);
				magz = mag*((float)rand()/RAND_MAX*2.0 - 1.0);
				
				force[i].x += magx;
				force[i].y += magy;
				force[i].z += magz;
			}
			else if(time < 1.5)
			{
				dragTemp = 0.01;
			}
			else
			{

				dragTemp = drag;

			}	

		}
		
		/*
		hipMemcpy( BodyPositionGPU, BodyPosition, NumberOfBodies*sizeof(float4), hipMemcpyHostToDevice );
		hipMemcpy( BodyVelocityGPU, BodyVelocity, NumberOfBodies*sizeof(float4), hipMemcpyHostToDevice );
		hipMemcpy( BodyForceGPU, BodyForce, NumberOfBodies*sizeof(float4), hipMemcpyHostToDevice );
		getForcesSetup<<<Grids, Blocks>>>(DevStates, BodyPositionGPU, BodyVelocityGPU, BodyForceGPU, PolymerConnectionAGPU, PolymerConnectionBGPU, PolymersConnectionLength, NumberOfPolymers, NumberOfMicroPlastics, BeakerRadius, FluidHeight, FluidDensity, StirFlag, Theta, ShakeItUpFlag);
		errorCheck("getForces");
		moveBodies<<<Grids, Blocks>>>(BodyPositionGPU, BodyVelocityGPU, BodyForceGPU, dragTemp, Dt, NumberOfBodies);
		errorCheck("moveBodies");
		hipMemcpy( BodyPosition, BodyPositionGPU, NumberOfBodies*sizeof(float4), hipMemcpyDeviceToHost );
		hipMemcpy( BodyVelocity, BodyVelocityGPU, NumberOfBodies*sizeof(float4), hipMemcpyDeviceToHost );
		hipMemcpy( BodyForce, BodyForceGPU, NumberOfBodies*sizeof(float4), hipMemcpyDeviceToHost );
		*/
		
		for(int i = 0; i < n; i++)
		{
			for(int j = i+1; j < n; j++)
			{
				dx = pos[j].x - pos[i].x;
				dy = pos[j].y - pos[i].y;
				dz = pos[j].z - pos[i].z;
				r2 = dx*dx + dy*dy + dz*dz + 0.000001;
				r = sqrt(r2);
				penitration = (pos[i].w + pos[j].w)/2.0 - r;
				
				if(0.0 < penitration)
				{
					// PolymerPolymer shell repulsion
					forceMag  = -k1*penitration*penitration;
				}
				else
				{
					// PolymerPolymer atraction
					forceMag  = 0.0;
				}
				force[i].x += forceMag*dx/r;
				force[i].y += forceMag*dy/r;
				force[i].z += forceMag*dz/r;
				
				force[j].x -= forceMag*dx/r;
				force[j].y -= forceMag*dy/r;
				force[j].z -= forceMag*dz/r;

				// Polymer chain connection forces.
				if(linkA[i] != -1 && j == linkA[i])
				{ 
					forceMag  = -k2*(length - r);
				}
				else if(linkB[i] != -1 && j == linkB[i])
				{ 
					forceMag  = -k2*(length - r);
				}
				force[i].x += forceMag*dx/r;
				force[i].y += forceMag*dy/r;
				force[i].z += forceMag*dz/r;
				
				force[j].x -= forceMag*dx/r;
				force[j].y -= forceMag*dy/r;
				force[j].z -= forceMag*dz/r;
			}
			
			dx = pos[i].x;
			dz = pos[i].z;
			r2 = dx*dx + dz*dz;
			r = sqrt(r2);
				
			if(beakerRadius < r)
			{
				forceMag = k*(beakerRadius - r);
				force[i].x = forceMag*pos[i].x/r;
				force[i].z = forceMag*pos[i].z/r;
			}
			if(fluidHeight < pos[i].y)
			{
				forceMag = k*(fluidHeight - pos[i].y);
				force[i].y = forceMag;
			}
			else if(pos[i].y < 0.0)
			{
				forceMag = -k*(pos[i].y);
				force[i].y = forceMag;
			}
		}
		
		for(int i = 0; i < n; i++)
		{	
			vel[i].x += ((force[i].x-dragTemp*vel[i].x)/force[i].w)*dt;
			vel[i].y += ((force[i].y-dragTemp*vel[i].y)/force[i].w)*dt;
			vel[i].z += ((force[i].z-dragTemp*vel[i].z)/force[i].w)*dt;

			pos[i].x += vel[i].x*dt;
			pos[i].y += vel[i].y*dt;
			pos[i].z += vel[i].z*dt;
		}
		
		time += dt;
	}
	
	printf("\n\n Polymers have been shoken up.");
	printf("\n");
}

void setInitailConditions()
{
	time_t t;
	srand((unsigned) time(&t));
	double density;
	double angle;
	double dx, dy, dz, d2, d;
	int k;
	int index;
	int test;
	double TotalPolymerLength;
	double spaceBetweenPolymerCenters;
	double startX,startY,startZ;
	
	// Zeroing out everything just for safety
	for(int i = 0; i < NumberOfBodies; i++)
	{
		BodyPosition[i].x = 0.0;
		BodyPosition[i].y = 0.0;
		BodyPosition[i].z = 0.0;
		BodyPosition[i].w = 0.0;
		
		BodyVelocity[i].x = 0.0;
		BodyVelocity[i].y = 0.0;
		BodyVelocity[i].z = 0.0;
		BodyVelocity[i].w = 0.0;
		
		BodyForce[i].x = 0.0;
		BodyForce[i].y = 0.0;
		BodyForce[i].z = 0.0;
		BodyForce[i].w = 0.0;
	}
	
	// Loading velocity, diameter, density, and mass of polymers
	for(int i = 0; i < NumberOfPolymers; i++)
	{
		BodyVelocity[i].x = 0.0;
		BodyVelocity[i].y = 0.0;
		BodyVelocity[i].z = 0.0;
		
		// Setting diameter
		BodyPosition[i].w = DiameterOfPolymer;	
		
		// Setting density
		BodyVelocity[i].w = DensityOfPolymer;
		
		// Setting mass
		BodyForce[i].w = DensityOfPolymer*(4.0/3.0)*PI*(BodyPosition[i].w/2.0)*(BodyPosition[i].w/2.0)*(BodyPosition[i].w/2.0);
	}
	
	// Setting velocity, diameter, density, and mass of microplastics
	for(int i = NumberOfPolymers; i < NumberOfBodies; i++)
	{
		BodyVelocity[i].x = 0.0;
		BodyVelocity[i].y = 0.0;
		BodyVelocity[i].z = 0.0;
		
		// Setting diameter
		BodyPosition[i].w = ((double)rand()/(double)RAND_MAX)*(DiameterOfMicroPlasticMax - DiameterOfMicroPlasticMin) + DiameterOfMicroPlasticMin;
		
		// Setting density
		density = ((double)rand()/(double)RAND_MAX)*(DensityOfMicroPlasticMax - DensityOfMicroPlasticMin) + DensityOfMicroPlasticMin;
		BodyVelocity[i].w = density;
		
		// Setting mass
		BodyForce[i].w = density*(4.0/3.0)*PI*(BodyPosition[i].w/2.0)*(BodyPosition[i].w/2.0)*(BodyPosition[i].w/2.0);	
	}
	
	// Setting intial positions of polymers
	spaceBetweenPolymerCenters = PolymersConnectionLength+DiameterOfPolymer;
	k = 0;
	for(int i = 0; i < NumberOfPolymerChains; i++)
	{
		test = 0;
		while(test == 0)
		{
			angle = 2.0*PI*(double)rand()/(double)RAND_MAX;
			BodyPosition[k].x = ((double)rand()/(double)RAND_MAX)*BeakerRadius * cos(angle);
			BodyPosition[k].z = ((double)rand()/(double)RAND_MAX)*BeakerRadius * sin(angle);
			
			test = 1;
			index = 0;
			for(int j = 0; j < i; j++)
			{
				// Checking against the leading element of the polymer chain.
				dx = BodyPosition[k].x - BodyPosition[index].x;
				dz = BodyPosition[k].z - BodyPosition[index].z;
				d2  = dx*dx + dz*dz;
				d = sqrt(d2); 
				if(d < spaceBetweenPolymerCenters)
				{
					test = 0;
				}
				index += PolymerChainLength[j];
			}
		}
		
		TotalPolymerLength = spaceBetweenPolymerCenters * (double)PolymerChainLength[i];
		BodyPosition[k].y = ((double)rand()/(double)RAND_MAX)*(FluidHeight - TotalPolymerLength) +  TotalPolymerLength;
		
		startX = BodyPosition[k].x;
		startY = BodyPosition[k].y;
		startZ = BodyPosition[k].z;
		
		PolymerConnectionA[k] = -1;
		PolymerConnectionB[k] = -1;
		k++;
		
		for(int j = 1; j < PolymerChainLength[i]; j++)
		{
			PolymerConnectionB[k-1] = k;
			PolymerConnectionA[k] = k-1;
			PolymerConnectionB[k] = -1;
			BodyPosition[k].x = startX;
			BodyPosition[k].y = startY - j*spaceBetweenPolymerCenters;
			BodyPosition[k].z = startZ;
			k++;
		}
	}
	
	if(DebugFlag == 1)
	{
		// Printing our polymer chains for debuging.
		k = 0;
		for(int i = 0; i < NumberOfPolymerChains; i++)
		{
			printf("\n ******************* polymer chain %d **********************\n", i);
			for(int j = 0; j < PolymerChainLength[i]; j++)
			{
				printf("PolymerPosition[%d] = (%f, %f, %f) linkA = %d linkB = %d \n", k, BodyPosition[k].x, BodyPosition[k].y, BodyPosition[k].z, PolymerConnectionA[k], PolymerConnectionB[k]);
				printf("PolymerVelocity[%d] = (%f, %f, %f) \n", k, BodyVelocity[k].x, BodyVelocity[k].y, BodyVelocity[k].z);
				k++;
			}
		}
	}
	
	// Shaking the polymers out of their unnatural intial positions.
	polymerShakeUp(BodyPosition, BodyVelocity, BodyForce, PolymerConnectionA, PolymerConnectionB, PolymersConnectionLength, NumberOfPolymers, Drag, Dt, BeakerRadius, FluidHeight);
	
	// Setting intial positions of micro plastics
	for(int i = NumberOfPolymers; i < NumberOfBodies; i++)
	{
		test = 0;
		while(test == 0)
		{
			angle = 2.0*PI*(double)rand()/(double)RAND_MAX;
			BodyPosition[i].x = ((double)rand()/(double)RAND_MAX)*(BeakerRadius * cos(angle));
			BodyPosition[i].y = ((double)rand()/(double)RAND_MAX)*(FluidHeight);
			BodyPosition[i].z = ((double)rand()/(double)RAND_MAX)*(BeakerRadius * sin(angle));
			
			test = 1;
			for(int j = 0; j < i; j++)
			{
				dx = BodyPosition[i].x - BodyPosition[j].x;
				dy = BodyPosition[i].y - BodyPosition[j].y;
				dz = BodyPosition[i].z - BodyPosition[j].z;
				d2  = dx*dx + dy*dy + dz*dz;
				d = sqrt(d2); 
				
				if(d < BodyPosition[i].w + BodyPosition[j].w)
				{
					test = 0;
				}
			}
		}
	}
	
	if(DebugFlag == 1)
	{
		// Printing micro plastics for debugging.
		printf("\n ****************************************** \n");
		for(int i = NumberOfPolymers; i < NumberOfBodies; i++)
		{
			printf(" MicrPlasticPosition[%d] = (%f, %f, %f) \n", i, BodyPosition[i].x, BodyPosition[i].y, BodyPosition[i].z);
		}
	}
		
	printf("\n\n Initial conditions have been set.");
	printf("\n");
}

void drawPicture()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	// Drawing Polymers
	for(int i = 0; i < NumberOfPolymers; i++)
	{
		glColor3d(PolymerRed, PolymerGreen, PolymerBlue);
		glPushMatrix();
			glTranslatef(BodyPosition[i].x, BodyPosition[i].y, BodyPosition[i].z);
			glutSolidSphere(BodyPosition[i].w/2.0, 30, 30);
		glPopMatrix();
		
		// Drawing polymer connections.
		// Note: there is no need to draw both. If you draw just the one above or below all 
		// connections will be drawn.
		glLineWidth(3.0);
		glColor3d(1.0, 0.0, 0.0);
		glBegin(GL_LINES);
			if(PolymerConnectionA[i] != -1)
			{
				glVertex3f(BodyPosition[i].x, BodyPosition[i].y, BodyPosition[i].z);
				glVertex3f(BodyPosition[PolymerConnectionA[i]].x, BodyPosition[PolymerConnectionA[i]].y, BodyPosition[PolymerConnectionA[i]].z);;
			}
		glEnd();
		
		if(DebugFlag == 1)
		{
			if(PolymerConnectionA[i] == -1)
			{
				glColor3d(0.0, 0.0, 1.0);
				glPushMatrix();
					glTranslatef(BodyPosition[i].x, BodyPosition[i].y, BodyPosition[i].z);
					glutSolidSphere(2.0*BodyPosition[i].w/2.0, 30, 30);
				glPopMatrix();
			}
			if(PolymerConnectionB[i] == -1)
			{
				glColor3d(0.0, 1.0, 1.0);
				glPushMatrix();
					glTranslatef(BodyPosition[i].x, BodyPosition[i].y, BodyPosition[i].z);
					glutSolidSphere(2.0*BodyPosition[i].w/2.0, 30, 30);
				glPopMatrix();
			}
		}
	}
	
	// Drawing Microplastics
	for(int i = NumberOfPolymers; i < NumberOfBodies; i++)
	{
		glColor3d(MicroPlasticRed, MicroPlasticGreen, MicroPlasticBlue);
		glPushMatrix();
			glTranslatef(BodyPosition[i].x, BodyPosition[i].y, BodyPosition[i].z);
			glutSolidSphere(BodyPosition[i].w/2.0, 30, 30);
		glPopMatrix();
	}
	
	// Drawint a red sphere at the origin for reference.
	glColor3d(1.0, 0.0, 0.0);
	glPushMatrix();
		glTranslatef(0, 0, 0);
		glutSolidSphere(10, 30, 30);
	glPopMatrix();
	
	// Drawing the outline of the Beaker.
	if(RadialConfinementViewingAids == 1)
	{
		glLineWidth(1.0);
		float divitions = 60.0;
		float angle = 2.0*PI/divitions;
		
		// Drawing top ring.
		glColor3d(0.0,1.0,0.0);
		for(int i = 0; i < divitions; i++)
		{
			glBegin(GL_LINES);
				glVertex3f(sin(angle*i)*BeakerRadius, FluidHeight, cos(angle*i)*BeakerRadius);
				glVertex3f(sin(angle*(i+1))*BeakerRadius, FluidHeight, cos(angle*(i+1))*BeakerRadius);
			glEnd();
		}

		glColor3d(0.0,1.0,0.0);
		for(int i = 0; i < divitions; i++)
		{
			glBegin(GL_LINES);
				glVertex3f(sin(angle*i)*BeakerRadius, 0.0, cos(angle*i)*BeakerRadius);
				glVertex3f(sin(angle*(i))*BeakerRadius, FluidHeight, cos(angle*(i))*BeakerRadius);
			glEnd();
		}
		
		// Drawing the bottom ring.
		glColor3d(1.0,1.0,1.0);
		for(int i = 0; i < divitions; i++)
		{
			glBegin(GL_LINES);
				glVertex3f(sin(angle*i)*BeakerRadius, 0.0, cos(angle*i)*BeakerRadius);
				glVertex3f(sin(angle*(i+1))*BeakerRadius, 0.0, cos(angle*(i+1))*BeakerRadius);
			glEnd();
		}
	}
	
	// Drawing the stirring.
	if(StirFlag == 1)
	{
		glLineWidth(2.0);
		glColor3d(1.0,0.0,0.0);
		glBegin(GL_LINES);
			glVertex3f(0.0, 0.0, 0.0);
			glVertex3f(BeakerRadius*cos(Theta), 0.0, BeakerRadius*sin(Theta));
		glEnd();
	}
	
	glutSwapBuffers();

	// Captures frames if you are making a movie.
	if(MovieFlag == 1)
	{
		glReadPixels(5, 5, XWindowSize, YWindowSize, GL_RGBA, GL_UNSIGNED_BYTE, Buffer);
		fwrite(Buffer, sizeof(int)*XWindowSize*YWindowSize, 1, MovieFile);
	}
}

/******************************************************************************
 This function initializes CUDA Rand making it so every thread can have its own set
 of random numbers.
*******************************************************************************/
__global__ void init_curand(unsigned int seed, hiprandState_t* states) 
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

/*
void brownian_motion(float3 *force)
{
	int i,under_normal_curve;
	float mag, angle1, angle2;
	float x,y,normal_hieght,temp;
	
	temp = 4.0*g_drag*DT;
	under_normal_curve = NO;
	
	while(under_normal_curve == NO)
	{
		x = 2.0*1.0*(float)rand()/RAND_MAX - 1.0;
		y = 1.0*(float)rand()/RAND_MAX;
		normal_hieght = 1.0*exp(-x*x/temp);
		if(y <= normal_hieght)
		{
			mag = x;
			under_normal_curve = YES;
		}
	}	
	
	for(i = 0; i < NUMBER_OF_BODIES; i++)
	{
		angle1 = PI*(float)rand()/RAND_MAX;
		angle2 = 2.0*PI*(float)rand()/RAND_MAX;
		force[i].x += mag*sinf(angle1)*cosf(angle2);
		force[i].y += mag*sinf(angle1)*sinf(angle2);
		force[i].z += mag*cosf(angle1);
	}
}
*/

/******************************************************************************
 This is the Brownian Motion function.
 Place any comments and papers you used to get parameters for this function here.
 The above commented out function is one I ased to get Brownian Motion in another project.
*******************************************************************************/
__device__ float4 brownian_motion(hiprandState_t* states, int id)
{
	float mag = 100.0;
	float4 f;
	float randx = mag*(hiprand_uniform(&states[id])*2.0 - 1.0);
        float randy = mag*(hiprand_uniform(&states[id])*2.0 - 1.0);
        float randz = mag*(hiprand_uniform(&states[id])*2.0 - 1.0);
        
        f.x = randx;
        f.y = randy;
        f.z = randz;
	
	return(f);
}

/******************************************************************************
 This function just shakes the whole system up
*******************************************************************************/
__device__ float4 shakeItUp(hiprandState_t* states, int id)
{
	float mag = 100.0;
	float4 v;
	float randx = mag*(hiprand_uniform(&states[id])*2.0 - 1.0);
        float randy = mag*(hiprand_uniform(&states[id])*2.0 - 1.0);
        float randz = mag*(hiprand_uniform(&states[id])*2.0 - 1.0);
        
        v.x = randx;
        v.y = randy;
        v.z = randz;
	
	return(v);
}

/******************************************************************************
 This is the Polymer to Polymer interaction function.
 Place any comments and papers you used to get parameters for this function here.
 
*******************************************************************************/                                 
__device__ float4 getPolymerPolymerForce(float4 p0, float4 p1, int linkA, int linkB, int yourId, float length, int myId)
{
    float4 f;
    float force;
    float dx = p1.x - p0.x;
    float dy = p1.y - p0.y;
    float dz = p1.z - p0.z;
    float r2 = dx*dx + dy*dy + dz*dz + 0.000001;
    float r = sqrt(r2);
    float penitration = (p0.w + p1.w)/2.0 - r;
    float k1 = 100.0;
    float k2 = 100.0;
    
    force  = 0.0;
    
    if(0.0 < penitration)
    {
    	// PolymerPolymer shell repulsion
    	force  += -k1*penitration*penitration;
    }
    else
    {
    	// PolymerPolymer atraction
    	force  += 0.0;
    }
    
    // Polymer chain connection forces.
    if(linkA != -1 && yourId == linkA)
    { 
    	force  += -k2*(length - r);
    }
    if(linkB != -1 && yourId == linkB)
    { 
    	force  += -k2*(length - r);
    }
    
    f.x = force*dx/r;
    f.y = force*dy/r;
    f.z = force*dz/r;
    
    return(f);
}

/******************************************************************************
 This is the Polymer to micro-plastic interaction function.
 Place any comments and papers you used to get parameters for this function here.
*******************************************************************************/
__device__ float4 getPolymerMicroPlasticForce(float4 p0, float4 p1)
{
    float4 f;
    float force;
    float dx = p1.x - p0.x;
    float dy = p1.y - p0.y;
    float dz = p1.z - p0.z;
    float r2 = dx*dx + dy*dy + dz*dz + 0.000001;
    float r = sqrt(r2);
    float G = 100.0;
    float penitration = (p0.w + p1.w)/2.0 - r;
    float k = 100.0;
    
    force  = 0.0;
 
    if(0.0 < penitration)
    {
    	// Polymer microPlastic shell repulsion.
    	force  += -k*penitration*penitration;
    }
    else
    {
    	// Polymer microPlastic actraction
    	force += G*(p0.w*p1.w)/r2;
    	//force += 0.0;
    	//printf("\n force = %f", force);
    }
    
    f.x = force*dx/r;
    f.y = force*dy/r;
    f.z = force*dz/r;
    return(f);
}

/******************************************************************************
 This is the micro-plasic to micro-plastic interaction function.
 Place any comments and papers you used to get parameters for this function here.
 Self-Assembled Plasmonic Nanoparticle Clusters: 
 https://www.science.org/doi/10.1126/science.1187949#editor-abstract
*******************************************************************************/
__device__ float4 getMicroPlasticMicroPlasticForce(float4 p0, float4 p1)
{
    float4 f;
    float force;
    float dx = p1.x - p0.x;
    float dy = p1.y - p0.y;
    float dz = p1.z - p0.z;
    float r2 = dx*dx + dy*dy + dz*dz + 0.000001;
    float r = sqrt(r2);
    float penitration = (p0.w + p1.w)/2.0 - r;
    float k = 100.0;
    
    force  = 0.0;
    
    if(0.0 < penitration)
    {
    	// MicroPlastic microPlastic shell repulsion.
    	force  += -k*penitration*penitration;
    }
    else
    {
    	// MicroPlastic microPlastic actraction
    	force  += 0.0;
    }
    
    f.x = force*dx/r;
    f.y = force*dy/r;
    f.z = force*dz/r;
    
    return(f);
}

/******************************************************************************
 This is the gravity function add complexity at will.
*******************************************************************************/
__device__ float4 getGravityForces(float density, float mass, float fluidDensity)
{
	float4 f;
	float G = 9.81; // When you take meters/second^2 to micrometers/millisecond^2 everything cancels out so you get 9.81
	f.x = 0.0;
	f.z = 0.0;
	
	// May want to do something more accurate. I just made a linear function that  pulled stuff dowm if its density is
	// greater than the fluid pushed it up if its density is less than the fluid.
	f.y = -G*mass*(density - fluidDensity);
	
	return(f);
}

/******************************************************************************
 This function keeps the bodies in the container.
*******************************************************************************/
__device__ float4 getContainerForces(float4 posMe, float beakerRadius, float fluidHeight)
{
	float4 f;
	float force;
	float r2 = posMe.x*posMe.x + posMe.z*posMe.z;
	float r = sqrt(r2);
	float k = 100.0;
	
	f.x = 0.0;
	f.y = 0.0;
	f.z = 0.0;
	
	if(beakerRadius < r)
	{
		force = k*(beakerRadius - r);
		f.x = force*posMe.x/r;
		f.z = force*posMe.z/r;
	}
	
	if(fluidHeight < posMe.y)
	{
		force = k*(fluidHeight - posMe.y);
		f.y = force;
	}
	else if(posMe.y < 0.0)
	{
		force = -k*(posMe.y);
		f.y = force;
	}
	
	return(f);
}

/******************************************************************************
 This is the stirring function add complexity at will.
*******************************************************************************/
__device__ float4 getStirringForces(hiprandState_t* states, int id, float4 posMe, float4 velMe, float beakerRadius, float fluidHeight, float theta)
{
	float4 f;
	float angle;
	float magRand = 1000.0;
	float centerMag;
	//float temp;
	float magStir = 20.0; 
	//float mag2 = 10.0;
	float r2 = posMe.x*posMe.x + posMe.z*posMe.z;
	float r = sqrt(r2);
	float range = PI/24.0;
	
	float randx = magRand*(hiprand_uniform(&states[id])*2.0 - 1.0);
        float randy = magRand*(hiprand_uniform(&states[id])*2.0 - 1.0);
        float randz = magRand*(hiprand_uniform(&states[id])*2.0 - 1.0);
	
	f.x = 0.0;
	f.y = 0.0;
	f.z = 0.0;
	
	if(0.0 < r)
	{
		// This gives a radial motion
		//f.x = mag1*(-posMe.z/r);
		//f.z = mag1*(posMe.x/r);
		
		// This gives a pulling down in the center and up on the sides.
		//f.y = mag2*(r*2.0/beakerRadius - 1.0);
		
		// This is suposed to move it in from the top and out on the bottom.
		//temp = 10.0*(1.0 - posMe.y/fluidHeight); //mag2*(-r*2.0/beakerRadius + 1.0);
		//f.x = temp*(posMe.x/r);
		//f.z = temp*(posMe.z/r);
		
		angle = atan(posMe.z/posMe.x);
		if(0.0 < posMe.x)
		{
			if(0.0 < posMe.z)
			{
				angle += 0.0;
			}
			else
			{
				angle += 2.0*PI;
			}
		}
		else
		{
			if(0.0 < posMe.z)
			{
				angle += PI;
			}
			else
			{
				angle += PI;
			}
		}
		
		if(0.0 < (angle - theta) < range)
		{
			centerMag = -(r/beakerRadius - 1.0)*(r/beakerRadius - 1.0) + 1.0; // This makes it full in the middle and die off on the ends,
			f.x = randx + centerMag*magStir*(-posMe.z/r);
			f.y = randy;
			f.z = randz + centerMag*magStir*(posMe.x/r);
		}
	}
	
	return(f);
}

__global__ void getForces(hiprandState_t* states, float4 *pos, float4 *vel, float4 *force, int *linkA, int *linkB, float length, int nPolymer, int nPlastics, float beakerRadius, float fluidHeight, float fluidDensity, int stirFlag, float theta, int shakeItUpFlag)
{
	int myId, yourId;
	int nBodies;
	float4 forceVector, forceVectorSum;
	float4 velocityVector;
	float4 posMe, posYou;
	float4 velMe;
	float densityMe, massMe;
	
	nBodies = nPolymer + nPlastics;
	myId = threadIdx.x + blockDim.x*blockIdx.x;
    	if(myId < nBodies)
    	{
		posMe.x = pos[myId].x;
		posMe.y = pos[myId].y;
		posMe.z = pos[myId].z;
		posMe.w = pos[myId].w;
		
		velMe.x = vel[myId].x;
		velMe.y = vel[myId].y;
		velMe.z = vel[myId].z;
		velMe.w = vel[myId].w;
		
		//diameterMe = posMe.w;
		densityMe = vel[myId].w;
		massMe = force[myId].w;
		
		forceVectorSum.x = 0.0;
		forceVectorSum.y = 0.0;
		forceVectorSum.z = 0.0;
		
		for(yourId = 0; yourId < nBodies; yourId++)
		{
			posYou.x = pos[yourId].x;
			posYou.y = pos[yourId].y;
			posYou.z = pos[yourId].z;
			posYou.w = pos[yourId].w;
			
			if(yourId != myId) // Making sure you are not working on youself.
			{
				if(myId < nPolymer)
				{
					if(yourId < nPolymer)
					{
						// Polymer-polymer force
						forceVector = getPolymerPolymerForce(posMe, posYou, linkA[myId], linkB[myId], yourId, length, myId);
					}
					else
					{
						// Polymer-microPlastic force
						forceVector = getPolymerMicroPlasticForce(posMe, posYou);
					}
				}
				else
				{
					if(yourId < nPolymer)
					{
						// Polymer-microPlastic force
						forceVector = getPolymerMicroPlasticForce(posMe, posYou);
					}
					else
					{
						// microPlastic-microPlastic force
						forceVector = getMicroPlasticMicroPlasticForce(posMe, posYou);
					}
				}
				
			    	forceVectorSum.x += forceVector.x;
			    	forceVectorSum.y += forceVector.y;
			    	forceVectorSum.z += forceVector.z;
		    	}
		}
		
		// This adds on a gravity pull based on density
		forceVector = getGravityForces(densityMe, massMe, fluidDensity);
		forceVectorSum.x += forceVector.x;
		forceVectorSum.y += forceVector.y;
		forceVectorSum.z += forceVector.z;
		
		// This adds on the forces to keep the bodies in the container.
		forceVector = getContainerForces(posMe, beakerRadius, fluidHeight);
		forceVectorSum.x += forceVector.x;
		forceVectorSum.y += forceVector.y;
		forceVectorSum.z += forceVector.z;
		
		// This adds on the forces caused by stirring.
		if(stirFlag == 1)
		{
			forceVector = getStirringForces(states, myId, posMe, velMe, beakerRadius, fluidHeight, theta);
			forceVectorSum.x += forceVector.x;
			forceVectorSum.y += forceVector.y;
			forceVectorSum.z += forceVector.z;
		}
		
		// This is adds Brownian Motion to the system.
		forceVector = brownian_motion(states, myId);
		forceVectorSum.x += forceVector.x;
		forceVectorSum.y += forceVector.y;
		forceVectorSum.z += forceVector.z;
		
		// This just adds random motion to the system.
		if(shakeItUpFlag == 1)
		{
			velocityVector = shakeItUp(states, myId);
			vel[myId].x += velocityVector.x;
			vel[myId].y += velocityVector.y;
			vel[myId].z += velocityVector.z;
		}
		
		// Tranfering all the forces to my force function
		force[myId].x = forceVectorSum.x;
		force[myId].y = forceVectorSum.y;
		force[myId].z = forceVectorSum.z;
    	}
}

__global__ void getForcesSetup(hiprandState_t* states, float4 *pos, float4 *vel, float4 *force, int *linkA, int *linkB, float length, int nPolymer, int nPlastics, float beakerRadius, float fluidHeight, float fluidDensity, int stirFlag, float theta, int shakeItUpFlag)
{
	int myId, yourId;
	int nBodies;
	float4 forceVector, forceVectorSum;
	float4 posMe, posYou;
	
	nBodies = nPolymer + nPlastics;
	myId = threadIdx.x + blockDim.x*blockIdx.x;
    	if(myId < nBodies)
    	{
		posMe.x = pos[myId].x;
		posMe.y = pos[myId].y;
		posMe.z = pos[myId].z;
		posMe.w = pos[myId].w;
		
		forceVectorSum.x = 0.0;
		forceVectorSum.y = 0.0;
		forceVectorSum.z = 0.0;
		
		for(yourId = 0; yourId < nBodies; yourId++)
		{
			posYou.x = pos[yourId].x;
			posYou.y = pos[yourId].y;
			posYou.z = pos[yourId].z;
			posYou.w = pos[yourId].w;
			
			if(yourId != myId) // Making sure you are not working on youself.
			{
				if(myId < nPolymer && yourId < nPolymer)
				{
					// Polymer-polymer force
					forceVector = getPolymerPolymerForce(posMe, posYou, linkA[myId], linkB[myId], yourId, length, myId);
				}
				
			    	forceVectorSum.x += forceVector.x;
			    	forceVectorSum.y += forceVector.y;
			    	forceVectorSum.z += forceVector.z;
		    	}
		}
		
		// This adds on the forces to keep the bodies in the container.
		forceVector = getContainerForces(posMe, beakerRadius, fluidHeight);
		forceVectorSum.x += forceVector.x;
		forceVectorSum.y += forceVector.y;
		forceVectorSum.z += forceVector.z;
		
		// This is adds Brownian Motion to the system.
		forceVector = brownian_motion(states, myId);
		forceVectorSum.x += forceVector.x;
		forceVectorSum.y += forceVector.y;
		forceVectorSum.z += forceVector.z;
		
		// Tranfering all the forces to my force function
		force[myId].x += forceVectorSum.x;
		force[myId].y += forceVectorSum.y;
		force[myId].z += forceVectorSum.z;
    	}
}


__global__ void moveBodies(float4 *pos, float4 *vel, float4 *force, float drag, float dt, int n)
{
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	if(id < n)
	{	
		vel[id].x += ((force[id].x-drag*vel[id].x)/force[id].w)*dt;
		vel[id].y += ((force[id].y-drag*vel[id].y)/force[id].w)*dt;
		vel[id].z += ((force[id].z-drag*vel[id].z)/force[id].w)*dt;

		pos[id].x += vel[id].x*dt;
		pos[id].y += vel[id].y*dt;
		pos[id].z += vel[id].z*dt;
	}
}

void nBody()
{
	if(Pause != 1)
	{	
		getForces<<<Grids, Blocks>>>(DevStates, BodyPositionGPU, BodyVelocityGPU, BodyForceGPU, PolymerConnectionAGPU, PolymerConnectionBGPU, PolymersConnectionLength, NumberOfPolymers, NumberOfMicroPlastics, BeakerRadius, FluidHeight, FluidDensity, StirFlag, Theta, ShakeItUpFlag);
		errorCheck("getForces");
		moveBodies<<<Grids, Blocks>>>(BodyPositionGPU, BodyVelocityGPU, BodyForceGPU, Drag, Dt, NumberOfBodies);
		errorCheck("moveBodies");
        	
        	DrawTimer++;
		if(DrawTimer == DrawRate) 
		{
			hipMemcpy( BodyPosition, BodyPositionGPU, NumberOfBodies*sizeof(float4), hipMemcpyDeviceToHost );
			drawPicture();
			//printf("\n Time = %f", RunTime);
			DrawTimer = 0;
		}
		
		PrintTimer++;
		if(PrintRate <= PrintTimer) 
		{
			terminalPrint();
			PrintTimer = 0;
			//printf("\n time = %f", RunTime);
		}
		
		RunTime += Dt; 
		if(TotalRunTime < RunTime)
		{
			printf("\n\n Done\n");
			exit(0);
		}
		
		Theta += StirAngularVelosity*Dt;
		if(2.0*PI < Theta) Theta = 0.0;
	}
}

void errorCheck(const char *message)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: %s = %s\n", message, hipGetErrorString(error));
		exit(0);
	}
}

void terminalPrint()
{
	if(DebugFlag != 1)
	{
		system("clear");
		//printf("\033[0;34m"); // blue.
		//printf("\033[0;36m"); // cyan
		//printf("\033[0;33m"); // yellow
		//printf("\033[0;31m"); // red
		//printf("\033[0;32m"); // green
		printf("\033[0m"); // back to white.
		
		printf("\n");
		printf("\033[0;33m");
		printf("\n **************************** Simulation Stats ****************************");
		printf("\033[0m");
		
		printf("\n Total run time = %7.2f milliseconds", RunTime);
		
		printf("\033[0;33m");
		printf("\n **************************** Terminal Comands ****************************");
		printf("\033[0m");
		//printf("\n h: Help");
		//printf("\n c: Recenter View");
		printf("\n c: Screenshot");
		//printf("\n k: Save Current Run");
		printf("\n");
		
		printf("\n Toggles");
		printf("\n r: Run/Pause            - ");
		if(Pause == 0) 
		{
			printf("\033[0;32m");
			printf(BOLD_ON "Simulation Running" BOLD_OFF);
		} 
		else
		{
			printf("\033[0;31m");
			printf(BOLD_ON "Simulation Paused" BOLD_OFF);
		}
		printf("\n v: Orthogonal/Frustum   - ");
		if (ViewFlag == 0) 
		{
			printf("\033[0;36m"); // cyan
			printf(BOLD_ON "Orthogonal" BOLD_OFF); 
		}
		else 
		{
			printf("\033[0;36m"); // cyan
			printf(BOLD_ON "Frustrum" BOLD_OFF);
		}
		printf("\n m: Video On/Off         - ");
		if (MovieFlag == 0) 
		{
			printf("\033[0;31m");
			printf(BOLD_ON "Video Recording Off" BOLD_OFF); 
		}
		else 
		{
			printf("\033[0;32m");
			printf(BOLD_ON "Video Recording On" BOLD_OFF);
		}
		printf("\n e: Radial Confinement Viewing Aid   - ");
		if(RadialConfinementViewingAids == 0) 
		{
			printf("\033[0;31m");
			printf(BOLD_ON "Radial Confinement Viewing Aid Off" BOLD_OFF);
		}
		else 
		{
			printf("\033[0;32m");
			printf(BOLD_ON "Radial Confinement Viewing Aid On" BOLD_OFF);
		}
		
		printf("\n");
		printf("\n Adjust views");
		printf("\n k/l: Rotate CW/CCW");
		printf("\n a/d: Translate Left/Right");
		printf("\n s/w: Translate Down/Up");
		printf("\n z/Z: Translate Out/In");
		printf("\n f:   Recenter");
		printf("\n");
		printf("\n ********************************************************************");
		printf("\033[0m");
		printf("\n");
	}
}

void setup()
{	
	readSimulationParameters();
	setNumberOfBodies();
	allocateMemory();
	setInitailConditions();
	hipMemcpy( BodyPositionGPU, BodyPosition, NumberOfBodies*sizeof(float4), hipMemcpyHostToDevice );
	hipMemcpy( BodyVelocityGPU, BodyVelocity, NumberOfBodies*sizeof(float4), hipMemcpyHostToDevice );
	hipMemcpy( BodyForceGPU, BodyForce, NumberOfBodies*sizeof(float4), hipMemcpyHostToDevice );
	hipMemcpy( PolymerConnectionAGPU, PolymerConnectionA, NumberOfPolymers*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( PolymerConnectionBGPU, PolymerConnectionB, NumberOfPolymers*sizeof(int), hipMemcpyHostToDevice );
	
	// Initialize CURAND
	//unsigned int seed = static_cast<unsigned int>(time(0));
    	init_curand<<<Grids, Blocks>>>(1234, DevStates);
    	errorCheck("init_curand");
	
	hipSetDevice(0); // Select GPU device 0
    	hipDeviceSynchronize();
    	errorCheck("hipSetDevice");
	
	DrawTimer = 0;
	PrintTimer = 0;
	RunTime = 0.0;
	Pause = 0;
	MovieFlag = 0;
	ViewFlag = 1;
	RadialConfinementViewingAids = 1;
	StirFlag = 1;
	ShakeItUpFlag = 0;
	DebugFlag = 0;
	Theta = 0.0;
	StirAngularVelosity = (2.0*PI)/(100.0); // This is 10 revolution per second in milliseconds
	
	CenterOfSimulation.x = 0.0;
	CenterOfSimulation.y = 0.0;
	CenterOfSimulation.z = 0.0;
	CenterOfSimulation.w = 0.0;
	
	AngleOfSimulation.x = 0.0;
	AngleOfSimulation.y = 1.0;
	AngleOfSimulation.z = 0.0;
	AngleOfSimulation.w = 0.0;
	
	terminalPrint();
}

	

int main(int argc, char** argv)
{
	setup();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 
	Buffer = new int[XWindowSize*YWindowSize];

	// Clip plains
	Near = 0.2;
	Far = BeakerRadius*6.0;

	//Direction here your eye is located location
	EyeX = 0.0;
	EyeY = FluidHeight+ 100;
	EyeZ = 1.0;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("N Body");
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	glClearColor(0.0, 0.0, 0.0, 0.0);
	
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutMouseFunc(mymouse);
	glutKeyboardFunc(KeyPressed);
	glutIdleFunc(idle);
	glutMainLoop();
	
	return 0;
}
